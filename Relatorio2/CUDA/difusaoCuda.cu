#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>


//#define N 10
#define N 2000  // Tamanho da grade
#define T 1000  // Quantidade de iterações
#define D 0.1   // Coeficiente de coesão
#define DELTA_T 0.01
#define DELTA_X 1.0

#define RADIUS 1
#define BLOCK_SIZE 16

#define cudaCheck(error) \
  if (error != hipSuccess) { \
    printf("Fatal error: %s at %s:%d\n", \
        hipGetErrorString(error), \
        __FILE__, __LINE__); \
    exit(1); \
}

__global__ void diff_eq(const double *input, double *output, int width, int height) {
    // Calculate thread indices - simplified for clarity
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int globalX = blockIdx.x * blockDim.x + tx;
    int globalY = blockIdx.y * blockDim.y + ty;

    // Shared memory for 2D block
    __shared__ double sharedMem[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

    // Local coordinates (including halo)
    int localX = tx + 1;
    int localY = ty + 1;

    // Initialize shared memory
    sharedMem[localY][localX] = 0.0;
    __syncthreads();

    // Load data into shared memory
    if (globalX < width && globalY < height) {
        sharedMem[localY][localX] = input[globalY * width + globalX];
    }

    // Load halo cells
    if (tx == 0 && globalX > 0) {
        sharedMem[localY][0] = input[globalY * width + (globalX - 1)];
    }
    if (tx == blockDim.x - 1 && globalX < width - 1) {
        sharedMem[localY][localX + 1] = input[globalY * width + (globalX + 1)];
    }
    if (ty == 0 && globalY > 0) {
        sharedMem[0][localX] = input[(globalY - 1) * width + globalX];
    }
    if (ty == blockDim.y - 1 && globalY < height - 1) {
        sharedMem[localY + 1][localX] = input[(globalY + 1) * width + globalX];
    }

    __syncthreads();

    // Compute stencil only for valid points
    if (globalX > 0 && globalX < width-1 && globalY > 0 && globalY < height-1) {
        double center = sharedMem[localY][localX];
        double north = sharedMem[localY-1][localX];
        double south = sharedMem[localY+1][localX];
        double west = sharedMem[localY][localX-1];
        double east = sharedMem[localY][localX+1];

        double newValue = center + D * (
            north + south + west + east - 4.0 * center
        ) * (DELTA_T / (DELTA_X * DELTA_X));

        output[globalY * width + globalX] = newValue; 
    }
}

__global__ void calculate_diffmedio(const double *input, const double *output, float *difmedio, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) { 
        float diff = fabs(output[y * width + x] - input[y * width + x]);
        atomicAdd(difmedio, diff);
    }
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Tamanho da matriz N*N (em bytes)
    size_t size = N * N * sizeof(double);

    // =========== Host setup =============

    // Alocar e inicializar a matriz no host
    double *host_C = (double *)malloc(size);
    double *host_C_output = (double *)malloc(size);

    // Verifica se a matriz foi criada corretamente
    if (host_C == NULL || host_C_output == NULL) {
      fprintf(stderr, "Falha na alocação de memória\n");
      return 1;
    }
    // Limpa a matrizes
    for (int i = 0; i < (N * N); i++) {
        host_C[i] = 0;
        host_C_output[i] = 0;
    }

    // Inicializa a concsentração no centro da matriz C
    host_C[((N/2) * N) + N/2] = 1.0;

    // ========== Device setup ============

    // Alocar e inicializar a matriz no device
    double *dev_C, *dev_C_output;

    cudaCheck(hipMalloc(&dev_C, size));
    cudaCheck(hipMalloc(&dev_C_output, size));

    // Verifica se a matriz foi criada corretamente
    if (dev_C == NULL || dev_C_output == NULL) {
        fprintf(stderr, "Failed to allocate device memory\n");
        free(host_C);
        free(host_C_output);
        return 1;
    }

    cudaCheck(hipMemcpy(dev_C, host_C, size, hipMemcpyHostToDevice));

    // Configura block size e threads
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                  (N + blockSize.y - 1) / blockSize.y);

    // Inicializa o tempo do código
    hipEventRecord(start); 

    // Executa o processo da equação de difusão com as matrizes
    for (int t = 0; t < T; t++) {
        diff_eq<<<gridSize, blockSize>>>(dev_C, dev_C_output, N, N);
        cudaCheck(hipDeviceSynchronize());

        // Calculate difference mean every 100 iterations
        if ((t % 100) == 0) {
          // Initialize difmedio on host and device
          float host_difmedio;
          float *dev_difmedio;

          hipMalloc(&dev_difmedio, sizeof(float));
          hipMemset(dev_difmedio, 0, sizeof(float)); 

          // Launch kernel to calculate difmedio
          calculate_diffmedio<<<gridSize, blockSize>>>(dev_C, dev_C_output, dev_difmedio, N, N);
          cudaCheck(hipDeviceSynchronize());

          // Copy difmedio from device to host
          hipMemcpy(&host_difmedio, dev_difmedio, sizeof(float), hipMemcpyDeviceToHost);

          // Calculate and print average difmedio
          host_difmedio /= ((N - 2) * (N - 2)); 
          printf("Iteração %d - diferença média=%g\n", t, host_difmedio);

          // Free device memory
          hipFree(dev_difmedio);
        }

        // Swap buffers
        double *temp = dev_C;
        dev_C = dev_C_output;
        dev_C_output = temp;
    }

    // Finaliza o tempo do processo da equação e salva-o
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Finaliza o tempo do processo da equação
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }
    // Synchronize and check for errors again
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    // Retornada dados ao hosst
    cudaCheck(hipMemcpy(host_C_output, dev_C_output, size, hipMemcpyDeviceToHost));

    printf("\nConcentração final no centro: %f\n", host_C_output[((N/2) * N) + N/2]);

    // Salvando matrix no aqruivo txt
    FILE *fp = fopen("/content/matriz_Cuda_output.txt", "w");

    if (fp == NULL) {
      printf("Erro ao abrir arquivo.txt\n");
    } else {
      for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
          if (host_C_output[i * N + j] >= 0.0001) {
            fprintf(fp, "i:%d j:%d Matriz:%f ", i, j, host_C_output[i * N + j]);
          }
        }
        fprintf(fp, "\n");
      }
      fclose(fp);
    }

    // Liberar memória alocada
    free(host_C);
    free(host_C_output);
    hipFree(dev_C);
    hipFree(dev_C_output);


    // Printa o tempo final
    printf("Tempo final do código: %f\n", elapsed_time / 1000.0);

    return 0;
}
